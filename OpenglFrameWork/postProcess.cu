#include "hip/hip_runtime.h"
#include"hip/device_functions.h"
#include ""
#include"hip/hip_vector_types.h"
#include<vector>
#include<cassert>
using std::vector;

//#define FILTER_WINDOW_RADIUS 3
//#define FILTER_WINDOW_DIAMETER 7
//#define FILTER_WINDOW_ELEMENT_COUNT 49


#define SIGMA 1
#define BLUR_DEPTH_IMPACT_FACTOR 0.2

surface<void, cudaSurfaceType2D> eyePosSurfaceRef;	//RGBA 32 F
surface<void, cudaSurfaceType2D> normalSurfaceRef;
surface<void, cudaSurfaceType2D> bluredEyePosSurfaceRef;

//__constant__ float filterWindow[FILTER_WINDOW_ELEMENT_COUNT];

hipChannelFormatDesc channelDesc;


hipArray * bluredEyePosTexMem;

__global__ void blurEyePosZKernel()
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 eyePos;
	float4 eyePosNear;
	surf2Dread(&eyePos, eyePosSurfaceRef, x * 16, y);
	if (eyePos.z > -0.001f) {
		surf2Dwrite(make_float4(0), bluredEyePosSurfaceRef, x * 16, y);
		return;
	}

	int filterRadius = __fdividef(480.0f, -eyePos.z);
	filterRadius = min(filterRadius, 10);
	float bluredZ = 0.0f;
	float sum = 0.0f;
	int tx;
	int ty; 
	float dDepth;
	float filterWeight;
	for (int i = -filterRadius; i <= filterRadius; ++i) {
		tx = x + i;
		if (tx < 0 || tx >= 800) {
			continue;
		}
		for (int j = -filterRadius; j <= filterRadius; ++j) {
			ty = y + j;
			if (ty < 0 || ty >= 800) {
				continue;
			}
			surf2Dread(&eyePosNear, eyePosSurfaceRef, tx * 16, ty);
			dDepth = abs(eyePos.z - eyePosNear.z) * BLUR_DEPTH_IMPACT_FACTOR;
			/*float weight = (exp(-0.5 * (pow((i - filterRadius) / SIGMA, 2.0) + pow((j - filterRadius) / SIGMA, 2.0)))
				/ (2 * 3.1415 * SIGMA * SIGMA)) * exp(-dDepth * dDepth);*/
			filterWeight = exp(-dDepth * dDepth);
			sum += filterWeight;
			bluredZ += (eyePosNear.z * filterWeight);
		}
	}
	
	eyePos.z = __fdividef(bluredZ, sum);
	surf2Dwrite(eyePos, bluredEyePosSurfaceRef, x * 16, y);
}
__global__ void calculateNormalKernel()
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	

	float4 eyePos;
	float4 eyePosNear;
	float4 dx;
	float4 dx2;
	float4 dy;
	float4 dy2;

	dx.z = dx2.z = dy.z = dy2.z = 999999.0f;

	surf2Dread(&eyePos, bluredEyePosSurfaceRef, x * 16, y);
	if (eyePos.z > -0.001f) {
		surf2Dwrite(make_float4(0), normalSurfaceRef, x * 16, y);
		return;
	}

	if (x != 0) {
		surf2Dread(&eyePosNear, bluredEyePosSurfaceRef, (x - 1) * 16, y);
		dx = eyePosNear - eyePos;
	}
	if (x < 799) {
		surf2Dread(&eyePosNear, bluredEyePosSurfaceRef, (x + 1) * 16, y);
		dx2 = eyePosNear - eyePos;
	}
	if (abs(dx.z) > abs(dx2.z)) {
		dx = dx2;
	}

	if (y != 0) {
		surf2Dread(&eyePosNear, bluredEyePosSurfaceRef, x * 16, y - 1);
		dy = eyePosNear - eyePos;
	}
	if (y < 799) {
		surf2Dread(&eyePosNear, bluredEyePosSurfaceRef, x * 16, y + 1);
		dy2 = eyePosNear - eyePos;
	}
	if (abs(dy.z) > abs(dy2.z)) {
		dy = dy2;
	}

	float3 normal = normalize(cross(make_float3(dx), make_float3(dy)));

	surf2Dwrite(make_float4(normal), normalSurfaceRef, x * 16, y);
}
void initPostProcess()
{
	channelDesc.x = 32;
	channelDesc.y = 32;
	channelDesc.z = 32;
	channelDesc.w = 32;
	channelDesc.f = hipChannelFormatKind::hipChannelFormatKindFloat;

	auto error = hipMallocArray(&bluredEyePosTexMem, &channelDesc, 800, 800,hipArraySurfaceLoadStore);
	assert(error == hipSuccess);



	//float filterWindowData[FILTER_WINDOW_ELEMENT_COUNT];
	//float weight = 1.0f / FILTER_WINDOW_ELEMENT_COUNT;

	//float sigma = 1;
	//float sum = 0.0;

	//int index = 0;
	//for (int i = 0; i < FILTER_WINDOW_DIAMETER; ++i) {
	//	for (int j = 0; j < FILTER_WINDOW_DIAMETER; ++j) {
	//		filterWindowData[index] = exp(-0.5 * (pow((i - FILTER_WINDOW_RADIUS) / sigma, 2.0) + pow((j - FILTER_WINDOW_RADIUS) / sigma, 2.0)))
	//			/ (2 * 3.1415 * sigma * sigma);
	//		sum += filterWindowData[index];
	//		++index;
	//	}
	//}
	//for (int i = 0; i < FILTER_WINDOW_ELEMENT_COUNT; ++i) {
	//	filterWindowData[i] /= sum;
	//}
	//hipMemcpyToSymbol(HIP_SYMBOL(filterWindow), filterWindowData, sizeof(float)* FILTER_WINDOW_ELEMENT_COUNT);
}

void invokePostProcessKernel(hipArray * eyePosArray,hipArray * normalArray)
{
	dim3 gridDim;
	dim3 blockDim;
	gridDim.x = gridDim.y = 25;
	gridDim.z = 1;

	blockDim.x = blockDim.y = 32;
	blockDim.z = 1;

	hipError_t error = cudaBindSurfaceToArray(&eyePosSurfaceRef, eyePosArray, &channelDesc);
	assert(error == hipSuccess);

	error = cudaBindSurfaceToArray(&normalSurfaceRef, normalArray, &channelDesc); 
	assert(error == hipSuccess);

	error = cudaBindSurfaceToArray(&bluredEyePosSurfaceRef, bluredEyePosTexMem, &channelDesc);
	assert(error == hipSuccess);

	blurEyePosZKernel << <gridDim, blockDim >> >();
	hipDeviceSynchronize();
	calculateNormalKernel << <gridDim, blockDim >> >();
}

void shutdownPostProcess()
{
	hipFreeArray(bluredEyePosTexMem);
}